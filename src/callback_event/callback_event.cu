#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <cupti.h>

#define CHECK_CUPTI_ERROR(err, cuptifunc)                                 \
    if (err != CUPTI_SUCCESS) {                                           \
        const char *errstr;                                               \
        cuptiGetResultString(err, &errstr);                               \
        fprintf(stderr, "%s:%d:Error %s for CUPTI API function '%s'.\n",  \
                __FILE__, __LINE__, errstr, cuptifunc);                   \
        exit(-1);                                                         \
    }

#define CHECK_CU_ERROR(err, cufunc)                                            \
    if (err != hipSuccess) {                                                 \
        fprintf(stderr, "%s:%d:Error %d for CUDA Driver API function '%s'.\n", \
                __FILE__, __LINE__, err, cufunc);                              \
        exit(-1);                                                              \
    }

// Structure to hold data collected by callback
typedef struct RuntimeApiTrace_st {
    CUpti_EventGroup eventGroup;
    CUpti_EventID eventId;
    uint64_t eventVal;
} RuntimeApiTrace_t;

// Device code
__global__ void vec_add(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

static void init_vec(int *vec, int n)
{
    int i;

    for (i = 0; i < n; i++)
        vec[i] = i;
}

static int run_vector_add()
{
    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int sum, i;
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    init_vec(h_A, N);
    init_vec(h_B, N);
    memset(h_C, 0, size);

    // Allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    vec_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (i = 0; i < N; ++i) {
        sum = h_A[i] + h_B[i];
        if (h_C[i] != sum) {
            printf("kernel execution FAILED\n");
            goto Error;
        }
    }

    return 0;
Error:
    return -1;
}

void CUPTIAPI getEventValueCallback(void *userdata, CUpti_CallbackDomain domain,
                                    CUpti_CallbackId cbid,
                                    const CUpti_CallbackData *cbInfo)
{
    CUptiResult cuptiErr;
    RuntimeApiTrace_t *traceData = (RuntimeApiTrace_t*)userdata;
    size_t bytesRead;

    // This callback is enabled only for launch so we shouldn't see anything else.
    if (cbid != CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) {
        printf("%s:%d: unexpected cbid %d\n", __FILE__, __LINE__, cbid);
        exit(-1);
    }

    if (cbInfo->callbackSite == CUPTI_API_ENTER) {
        hipDeviceSynchronize();
        cuptiErr = cuptiSetEventCollectionMode(cbInfo->context, 
                CUPTI_EVENT_COLLECTION_MODE_KERNEL);
        CHECK_CUPTI_ERROR(cuptiErr, "cuptiSetEventCollectionMode");
        cuptiErr = cuptiEventGroupEnable(traceData->eventGroup);
        CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupEnable");
    }

    if (cbInfo->callbackSite == CUPTI_API_EXIT) {
        bytesRead = sizeof (uint64_t);
        hipDeviceSynchronize();
        cuptiErr = cuptiEventGroupReadEvent(traceData->eventGroup, 
                CUPTI_EVENT_READ_FLAG_FORCE_INT, 
                traceData->eventId, 
                &bytesRead, &traceData->eventVal);
        CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupReadEvent");

        cuptiErr = cuptiEventGroupDisable(traceData->eventGroup);
        CHECK_CUPTI_ERROR(cuptiErr, "cuptiEventGroupDisable");
    }
}

static void cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
}

static uint64_t cupti_profile_event(hipDevice_t dev, CUpti_EventID event_id)
{
    CUpti_SubscriberHandle subscriber;
    CUpti_EventGroup event_group;
    RuntimeApiTrace_t trace;
    CUptiResult cupti_res;
    hipCtx_t context;
    hipError_t cu_res;

    // Create a new CUDA context and associates it with the calling thread.
    cu_res = hipCtxCreate(&context, 0, dev);
    CHECK_CU_ERROR(cu_res, "hipCtxCreate");

    // Create a new event group for a context.
    cupti_res = cuptiEventGroupCreate(context, &event_group, 0);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiEventGroupCreate");

    // Add an event to an event group.
    cupti_res = cuptiEventGroupAddEvent(event_group, event_id);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiEventGroupAddEvent");

    // Configure trace.
    trace.eventGroup = event_group;
    trace.eventId    = event_id;

    // Initialize a callback subscriber with a callback function and user data.
    cupti_res = cuptiSubscribe(&subscriber,
            (CUpti_CallbackFunc)getEventValueCallback,
            &trace);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiSubscribe");

    // Enable or disabled callbacks for a specific domain and callback ID.
    cupti_res = cuptiEnableCallback(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API,
            CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiEnableCallback");

    // Run a CUDA sample.
    run_vector_add();

    // Unregister a callback subscriber.
    cupti_res = cuptiUnsubscribe(subscriber);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiUnsubscribe");

    // Remove an event from an event group.
    cupti_res = cuptiEventGroupRemoveEvent(event_group, event_id);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiEventGroupRemoveEvent");

    // Destroy an event group.
    cupti_res = cuptiEventGroupDestroy(event_group);
    CHECK_CUPTI_ERROR(cupti_res, "cuptiEventGroupDestroy");

    // Destroy the CUDA context.
    cu_res = hipCtxDestroy(context);
    CHECK_CU_ERROR(cu_res, "hipCtxDestroy");

    return trace.eventVal;
}

int main(int argc, char **argv)
{
    CUptiResult cupti_ret = CUPTI_SUCCESS;
    CUpti_EventID event_id;
    uint64_t event_val;
    int device_id = 0; /* Assuming device 0 by default */
    char *event_name;
    hipDevice_t dev = 0;
    int device_count;
    hipError_t ret;

    if (argc < 2) {
        fprintf(stderr, "Usage: %s <event_name> [device_num]\n", argv[0]);
        return -1;
    }
    event_name = argv[1];

    if (argc > 2)
        device_id = atoi(argv[2]);

    ret = hipInit(0);
    CHECK_CU_ERROR(ret, "hipInit");

    ret = hipGetDeviceCount(&device_count);
    CHECK_CU_ERROR(ret, "hipGetDeviceCount");

    if (device_count == 0) {
        fprintf(stderr, "There is no device supporting CUDA.\n");
        return -1;
    }

    ret = hipDeviceGet(&dev, device_id);
    CHECK_CU_ERROR(ret, "hipDeviceGet");

    cupti_ret = cuptiEventGetIdFromName(dev, event_name, &event_id);
    CHECK_CUPTI_ERROR(cupti_ret, "cuptiEventGetIdFromName");

    event_val = cupti_profile_event(dev, event_id);

    printf("Event Name : %s \n",   event_name);
    printf("Event Value : %llu\n", (unsigned long long)event_val);

    hipDeviceSynchronize();
    return 0;
}

